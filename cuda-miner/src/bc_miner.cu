#include "hip/hip_runtime.h"
// The basic components of a GPU-based block collider miner
// lgray@github September 2018
// permission granted to use under MIT license
// this is a GPU miner for block collider that does ~ 20M hashes + distances per second

#include "bc_miner.h"
#include "blake2.h"
#include "blake2b.cu"
#include "cos_dist.cu"
#include <hiprand/hiprand_kernel.h>
#include "stdio.h"
#include <random>
#include <chrono>
#include <pthread.h>

//mutexes
pthread_mutex_t solution_found_mutex = PTHREAD_MUTEX_INITIALIZER;
pthread_mutex_t data_xfer_mutex = PTHREAD_MUTEX_INITIALIZER;

__global__ void setup_rand(hiprandState* state, uint32_t random)
{
  unsigned id = threadIdx.x + blockIdx.x * blockDim.x;
  unsigned clk = (unsigned)clock64();  
  /* Each thread gets same seed, a different sequence 
     number, no offset */
  hiprand_init(id+random +clk, 0, 0, &state[id]);
}

//__device__ __host__ __forceinline__ 
__global__
void one_unit_work(bc_mining_data* mining_info) {
  
  unsigned id = threadIdx.x + blockIdx.x *blockDim.x;
  
  uint8_t data_in[bc_mining_data::INLENGTH];
  //memset(data_in,0,bc_mining_data::INLENGTH); // this memset is unecessary 
  
  const size_t idoffset = id*BLAKE2B_OUTBYTES;
  memcpy(data_in,mining_info->work_template_,mining_info->work_size_);
  memcpy(data_in+mining_info->nonce_hash_offset_,mining_info->nonce_hashes+idoffset,BLAKE2B_OUTBYTES);

  
  blake2b_state s;
  blake2b_init_cu(&s,BLAKE2B_OUTBYTES);  
  blake2b_update_cu(&s,data_in,mining_info->work_size_);
  blake2b_final_cu(&s,mining_info->result+idoffset,BLAKE2B_OUTBYTES);
  

  mining_info->distance[id] = cosine_distance_cu(mining_info->received_work_,
						 mining_info->result+id*BLAKE2B_OUTBYTES);
}

__global__
void prepare_work_nonces(hiprandState *state, uint64_t startnonce, bc_mining_data* mining_info) {

  static uint16_t num_to_code[16] =  {48,49,50,51,52,53,54,55,56,57,97,98,99,100,101,102};  
  static uint8_t nonce_prefix[28] = "/WAIETING/HOODIE/WITHOUTIRE/";

  unsigned id = threadIdx.x + blockIdx.x * blockDim.x;
    
  hiprandState localState = state[id];
  uint8_t nonce_string[NONCESIZE]; // up to 64 bytes of nonce
  uint8_t nonce_hash[BLAKE2B_OUTBYTES];
  memset(nonce_string,0,NONCESIZE);

  //2060688607;
  uint64_t nonce = startnonce + id + hiprand(&localState) + ( ((uint64_t)hiprand(&localState)) << 32 );
  
  // convert nonce
  nonce_string[0] = '0'; // take care of base case
  uint32_t length = 0;
  uint64_t red_nonce = nonce;
  while( red_nonce > 0 ) { ++length; red_nonce /= 10ULL; }
  red_nonce = nonce;
  for( uint64_t i = length; i > 1; --i ) {
    nonce_string[i-1] = num_to_code[red_nonce%10];
    red_nonce /= 10ULL;
  }
  nonce_string[0] = num_to_code[red_nonce];
  length = (length == 0) + (length > 0)*length;
  memcpy(nonce_string + length, nonce_prefix, 28);
  length += 28;

  //printf("length: %u %llu %s\n",length,nonce,nonce_string); 
  
  // create the nonce hash
  blake2b_state ns;
  blake2b_init_cu(&ns,BLAKE2B_OUTBYTES);  
  blake2b_update_cu(&ns,nonce_string,length);
  blake2b_final_cu(&ns,nonce_hash,BLAKE2B_OUTBYTES);

  // hash the hash for extra hashiness
  //blake2b_state ns1;
  //blake2b_init_cu(&ns1,BLAKE2B_OUTBYTES);
  //blake2b_update_cu(&ns1,nonce_hash,BLAKE2B_OUTBYTES);
  //blake2b_final_cu(&ns1,nonce_hash_hash,BLAKE2B_OUTBYTES);

  // convert nonce in place to string codes and "blake2bl" form
  #pragma unroll
  for( unsigned i = 32; i < BLAKE2B_OUTBYTES; ++i ) {
    uint8_t byte = nonce_hash[i];
    nonce_hash[2*(i-32)] = num_to_code[byte>>4];
    nonce_hash[2*(i-32)+1] = num_to_code[byte&0xf];
  }
    
  // now we put everything into the data_in string in stringified hex form  
  const size_t idoffset = id*BLAKE2B_OUTBYTES;
  memcpy(mining_info->nonce_hashes+idoffset,
	 nonce_hash,
	 BLAKE2B_OUTBYTES);  

  //copy the local work back to the gpu memory  
  memcpy(mining_info->nonce + id*NONCESIZE, nonce_string, length);

  state[id] = localState;
}

__global__ void prepare_max_distance(uint64_t *max, uint64_t *maxidx, const uint64_t *a) {
  __shared__ uint64_t maxtile[N_MINER_THREADS_PER_BLOCK];
  __shared__ uint64_t maxidxtile[N_MINER_THREADS_PER_BLOCK];
  
  unsigned int tid = threadIdx.x;
  uint64_t i = blockIdx.x * blockDim.x + threadIdx.x;
  maxtile[tid] = a[i];
  maxidxtile[tid] = i;
  __syncthreads();
  
  //sequential addressing by reverse loop and thread-id based indexing
  for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (tid < s) {
      if (maxtile[tid + s] > maxtile[tid]) {
	maxtile[tid] = maxtile[tid + s];
	maxidxtile[tid] = maxidxtile[tid + s];
      }
    }
    __syncthreads();
  }
  
  if (tid == 0) {
    max[blockIdx.x] = maxtile[0];
    maxidx[blockIdx.x] = maxidxtile[0];
  }
}

__global__ void finalize_max_distance(uint64_t *max, uint64_t *maxidx) {
  __shared__ uint64_t maxtile[N_MINER_THREADS_PER_BLOCK];
  __shared__ uint64_t maxidxtile[N_MINER_THREADS_PER_BLOCK];

  unsigned int tid = threadIdx.x;
  uint64_t i = blockIdx.x * blockDim.x + threadIdx.x;
  maxtile[tid] = max[i];
  maxidxtile[tid] = maxidx[i];
  __syncthreads();
  
  //sequential addressing by reverse loop and thread-id based indexing
  for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (tid < s) {
      if (maxtile[tid + s] > maxtile[tid]) {
	maxtile[tid] = maxtile[tid + s];
	maxidxtile[tid] = maxidxtile[tid + s];
      }
    }
    __syncthreads();
  }
  
  if (tid == 0) {    
    max[blockIdx.x] = maxtile[0];
    maxidx[blockIdx.x] = maxidxtile[0];
  }
}

void init_gpus(std::vector<bc_mining_stream>& streams) {
  streams.clear();
  int nGPUs = 0;
  hipGetDeviceCount(&nGPUs);
  std::cout << "Found " << nGPUs << " GPUs to use for mining!" << std::endl;

  streams.resize(nGPUs);
  for( unsigned iGPU = 0; iGPU < nGPUs; ++iGPU ) {
    streams[iGPU].device = iGPU;
    hipSetDevice(iGPU);
    hipDeviceReset();
    hipStreamCreate(&streams[iGPU].stream);
    init_mining_memory(streams[iGPU].pool,streams[iGPU].stream);
  }
}

// create the primary mining work areas
// run this once to create the memory pools necessary for mining
// large cudaMallocs take a long time, hipMemset is fast
void init_mining_memory(bc_mining_mempools& pool, hipStream_t stream) {
  if( pool.dev_cache != NULL ) return;
  if( pool.dev_states != NULL ) return;
  if( pool.scratch_dists != NULL ) return;
  if( pool.scratch_indices != NULL ) return;

  // allocate device memory for random states and hashing work
  hipStreamSynchronize(stream);
  hipMalloc((void **)&pool.dev_states, HASH_TRIES * 1 * sizeof(hiprandState));
  hipMalloc(&pool.dev_cache,sizeof(bc_mining_data));
  hipMalloc(&pool.scratch_dists,HASH_TRIES*sizeof(uint64_t));
  hipMalloc(&pool.scratch_indices,HASH_TRIES*sizeof(uint64_t));
  hipStreamSynchronize(stream);
}

void run_miner(const bc_mining_inputs& in, const uint64_t start_nonce, bc_mining_stream& bcstream, bc_mining_outputs& out, bool& solution_found, bool& cancel) {
  hipSetDevice(bcstream.device);
  hipStream_t stream = bcstream.stream;
  bc_mining_mempools& pool = bcstream.pool;

  if( pool.dev_cache == NULL ) return;
  if( pool.dev_states == NULL ) return;
  if( pool.scratch_dists == NULL ) return;
  if( pool.scratch_indices == NULL ) return;
  
  uint64_t nonce_local = start_nonce;

  unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
  std::mt19937_64 generator(seed);

  dim3 threads(N_MINER_THREADS_PER_BLOCK,1,1), blocks(HASH_TRIES/N_MINER_THREADS_PER_BLOCK,1,1);
  
  //random numbers
  uint16_t work_size = in.miner_key_size_ + 2*BLAKE2B_OUTBYTES + in.time_stamp_size_;
  uint16_t nonce_hash_offset = in.miner_key_size_ + BLAKE2B_OUTBYTES;

  // prepare the mining work
  hipMemsetAsync(pool.dev_cache,0,sizeof(bc_mining_data),stream);
  hipMemcpyAsync(&pool.dev_cache->time_stamp_size_, &in.time_stamp_size_, sizeof(size_t), hipMemcpyHostToDevice,stream);
  hipMemcpyAsync(pool.dev_cache->time_stamp_, in.time_stamp_, in.time_stamp_size_, hipMemcpyHostToDevice,stream);
  hipMemcpyAsync(&pool.dev_cache->miner_key_size_, &in.miner_key_size_, sizeof(size_t), hipMemcpyHostToDevice,stream);
  hipMemcpyAsync(pool.dev_cache->miner_key_, in.miner_key_, in.miner_key_size_, hipMemcpyHostToDevice,stream);
  hipMemcpyAsync(pool.dev_cache->received_work_, in.received_work_, BLAKE2B_OUTBYTES, hipMemcpyHostToDevice,stream);
  hipMemcpyAsync(pool.dev_cache->merkel_root_,in.merkel_root_, BLAKE2B_OUTBYTES, hipMemcpyHostToDevice,stream);

  //setup the work template
  hipMemsetAsync(pool.dev_cache->work_template_,0,bc_mining_data::INLENGTH,stream);
  hipMemcpyAsync(&pool.dev_cache->nonce_hash_offset_,&nonce_hash_offset,sizeof(uint16_t),hipMemcpyHostToDevice,stream);
  hipMemcpyAsync(&pool.dev_cache->work_size_,&work_size,sizeof(uint16_t),hipMemcpyHostToDevice,stream);
  unsigned index = 0;
  hipMemcpyAsync(pool.dev_cache->work_template_,pool.dev_cache->miner_key_,in.miner_key_size_,hipMemcpyDeviceToDevice,stream);
  index += in.miner_key_size_;
  hipMemcpyAsync(pool.dev_cache->work_template_+index,pool.dev_cache->merkel_root_,BLAKE2B_OUTBYTES,hipMemcpyDeviceToDevice,stream);
  index += 2*BLAKE2B_OUTBYTES; //advance past nonce hash area
  hipMemcpyAsync(pool.dev_cache->work_template_+index,pool.dev_cache->time_stamp_,in.time_stamp_size_,hipMemcpyDeviceToDevice,stream);
  index += in.time_stamp_size_;
  
  // work areas for finding max
  uint64_t best_value(0);
  uint64_t max_value(0), max_idx(0);
  hipMemsetAsync(pool.scratch_dists,0,HASH_TRIES*sizeof(uint64_t),stream);
  hipMemsetAsync(pool.scratch_indices,0,HASH_TRIES*sizeof(uint64_t),stream);
  
  uint64_t iterations = 0;
  // the following kernel launches are the primary work
  // only set the random seeds once
  
  setup_rand<<<blocks,threads,0,stream>>>(pool.dev_states,((const uint32_t*)in.received_work_)[0]^((uint32_t)start_nonce));
  do {
    //hipMemsetAsync(pool.dev_states,0,HASH_TRIES*sizeof(hiprandState),stream);
    //setup_rand<<<blocks,threads,0,stream>>>(pool.dev_states,((const uint32_t*)in.received_work_)[0]^((uint32_t)nonce_local));

    if( solution_found || cancel ) break;
    hipMemsetAsync(pool.dev_cache->result,0,HASH_TRIES*BLAKE2B_OUTBYTES,stream);
    hipMemsetAsync(pool.dev_cache->nonce,0,HASH_TRIES*NONCESIZE,stream);
    hipMemsetAsync(pool.dev_cache->nonce_hashes,0,HASH_TRIES*BLAKE2B_OUTBYTES,stream);
   
    prepare_work_nonces<<<blocks,threads,0,stream>>>(pool.dev_states, nonce_local, pool.dev_cache);
    one_unit_work<<<blocks,threads,0,stream>>>(pool.dev_cache);
    hipMemsetAsync(pool.scratch_dists,0,HASH_TRIES*sizeof(uint64_t),stream);
    hipMemsetAsync(pool.scratch_indices,0,HASH_TRIES*sizeof(uint64_t),stream);
    prepare_max_distance<<<blocks,threads,0,stream>>>(pool.scratch_dists,pool.scratch_indices,pool.dev_cache->distance);
    unsigned temp = blocks.x;
    while( temp > threads.x ) {
      temp /= threads.x;
      finalize_max_distance<<<temp,threads,0,stream>>>(pool.scratch_dists,pool.scratch_indices);
    }
    finalize_max_distance<<<1,temp,0,stream>>>(pool.scratch_dists,pool.scratch_indices);
    // get the max value and index, which are at index zero in the scratch arrays
    hipMemcpyAsync(&max_value,pool.scratch_dists,sizeof(uint64_t),hipMemcpyDeviceToHost,stream);
    hipMemcpyAsync(&max_idx,pool.scratch_indices,sizeof(uint64_t),hipMemcpyDeviceToHost,stream);
    hipStreamSynchronize(stream);
    if( max_value > best_value ) {
      best_value = max_value;
      const uint64_t offsetb2b = max_idx*BLAKE2B_OUTBYTES;
      hipMemcpyAsync(out.result_blake2b_,pool.dev_cache->result+offsetb2b, BLAKE2B_OUTBYTES,hipMemcpyDeviceToHost,stream);
      hipMemcpyAsync(&out.nonce_, pool.dev_cache->nonce + max_idx*NONCESIZE, NONCESIZE, hipMemcpyDeviceToHost,stream);
    }
    ++iterations;    
    nonce_local = generator() ^ generator();
  } while( max_value <= in.the_difficulty_ && !solution_found && !cancel);

  if( !cancel ) {
    std::cout << bcstream.device << " found solution! " << max_value << std::endl;
    pthread_mutex_lock( &solution_found_mutex );
    if( !solution_found ) solution_found = true;
    pthread_mutex_unlock( &solution_found_mutex );

    out.difficulty_ = in.the_difficulty_;
    out.distance_ = best_value;
    out.iterations_ = iterations*HASH_TRIES;
    out.canceled_ = false;
  } else {
    std::cout << bcstream.device << " canceled!" << std::endl;

    out.difficulty_ = in.the_difficulty_;
    out.distance_ = best_value;
    out.iterations_ = iterations*HASH_TRIES;
    out.canceled_ = true;
  }

}

void* run_miner_thread(void * input) {
  bc_thread_data& inputs = *((bc_thread_data*)input);
  run_miner(*inputs.in,inputs.start_nonce,*inputs.stream,*inputs.out, *inputs.solution_found, *inputs.cancel);
  return NULL;
}

void destroy_mining_memory(bc_mining_mempools& pool, hipStream_t stream) {
  if( pool.dev_cache == NULL ) return;
  if( pool.dev_states == NULL ) return;
  if( pool.scratch_dists == NULL ) return;
  if( pool.scratch_indices == NULL ) return;

  // free device memory
  hipStreamSynchronize(stream);
  hipFree(pool.dev_states);
  hipFree(pool.dev_cache);
  hipFree(pool.scratch_dists);
  hipFree(pool.scratch_indices);
  hipStreamSynchronize(stream);

  // set it to null
  pool.dev_states = NULL;
  pool.dev_cache = NULL;
  pool.scratch_dists = NULL;
  pool.scratch_indices = NULL;
}

void destroy_gpus(std::vector<bc_mining_stream>& streams) {
  for(unsigned i = 0; i < streams.size(); ++i ) {
    hipSetDevice(streams[i].device);
    destroy_mining_memory(streams[i].pool,streams[i].stream);
    hipStreamDestroy(streams[i].stream);
  }
  streams.resize(0);
}
